#include "hip/hip_runtime.h"
#include <cstddef>
#include <stdio.h>
#include "mpi.h"
#include "nvshmem.h"
#include "nvshmemx.h"
#include <iostream>
#include <assert.h>
#define MAX_MSG_SIZE (32 * 1024 * 1024)

#define MAX_ITERS 200
#define MAX_SKIP 20
#define BLOCKS 4
#define THREADS_PER_BLOCK 1024

#define CUDA_CHECK(stmt)                                  \
do {                                                      \
    hipError_t result = (stmt);                          \
    if (hipSuccess != result) {                          \
        fprintf(stderr, "[%s:%d] CUDA failed with %s \n", \
         __FILE__, __LINE__, hipGetErrorString(result)); \
        exit(-1);                                         \
    }                                                     \
} while (0)

const int message_size = 64;

__global__ void simple_shift(int *destination) {
    int mype = nvshmem_my_pe();
    int npes = nvshmem_n_pes();
    int peer = (mype + 1) % npes;

    // nvshmem_int_p(destination, mype, peer);
    nvshmemx_int_put_nbi_block(destination,destination, message_size, peer);
}

int main (int argc, char *argv[]) {
    int mype_node;
    int* msg = (int*)malloc(sizeof(int)*message_size);
    hipStream_t stream;
    int rank, nranks;
    MPI_Comm mpi_comm = MPI_COMM_WORLD;
    nvshmemx_init_attr_t attr;
    std::cout<<"complete nvshmemx_init_attr_t attr"<<std::endl;
    MPI_Init(&argc, &argv);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);
    MPI_Comm_size(MPI_COMM_WORLD, &nranks);
    std::cout<<"complete MPI_Init"<<std::endl;
    attr.mpi_comm = &mpi_comm;
    nvshmemx_init_attr(NVSHMEMX_INIT_WITH_MPI_COMM, &attr);
    mype_node = nvshmem_team_my_pe(NVSHMEMX_TEAM_NODE);
    std::cout<<"complete nvshmemx_init_attr"<<std::endl;
    CUDA_CHECK(hipSetDevice(mype_node));
    CUDA_CHECK(hipStreamCreate(&stream));
    std::cout<<"complete hipSetDevice"<<std::endl;
    int *destination = (int *) nvshmem_malloc (sizeof(int)*message_size);
    for(int i=0; i<message_size; i++) {
        destination[i] = i;
    }
    std::cout<<"complete nvshmem_malloc"<<std::endl;
    simple_shift<<<1, 1, 0, stream>>>(destination);
    std::cout<<"complete simple_shift"<<std::endl;
    nvshmemx_barrier_all_on_stream(stream);
    std::cout<<"complete nvshmemx_barrier_all_on_stream"<<std::endl;
    CUDA_CHECK(hipMemcpyAsync(msg, destination, sizeof(int)*message_size,
                hipMemcpyDeviceToHost, stream));
    std::cout<<"complete hipMemcpyAsync"<<std::endl;
    CUDA_CHECK(hipStreamSynchronize(stream));
    std::cout<<"complete hipStreamSynchronize"<<std::endl;
    printf("%d: received message %d\n", nvshmem_my_pe(), *msg);

    nvshmem_free(destination);
    nvshmem_finalize();
    MPI_Finalize();
    return 0;
}
